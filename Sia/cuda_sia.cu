#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <cuda_helper.h>
#include "balloon.h"
#include "../sha256-sse/sha256.h"
#include "miner.h"
#include "../neoscrypt/cuda_vector_tpruvot.cuh"

static uint32_t *d_KNonce2[MAX_GPUS];
__global__ void conv_onethread(int n, int fn, const float * signal, const float * filter, float * retSignal);
__device__ void cuda_hash_state_mix(struct hash_state *s, int32_t mixrounds, uint64_t *prebuf_le);
__device__ void device_sha256_osol(const __sha256_block_t blk, __sha256_hash_t ctx);
__device__ void device_sha256_168byte(uint8_t *data, uint8_t *outhash);
__device__ void cuda_hash_state_extract(const struct hash_state *s, uint8_t out[BLOCK_SIZE]);
__device__ void cuda_compress(uint64_t *counter, uint8_t *out, const uint8_t *blocks[], size_t blocks_to_comp);
__device__ void cuda_expand(uint64_t *counter, uint8_t *buf, size_t blocks_in_buf);
__device__ void cuda_hash_state_fill(struct hash_state *s, const uint8_t *in, size_t inlen, int32_t t_cost, int64_t s_cost);
__device__ void device_sha256_generic(uint8_t *data, uint8_t *outhash, uint32_t len);
void host_sha256_osol(const __sha256_block_t blk, __sha256_hash_t ctx);
__global__ void cudaized_multi(struct hash_state *s, int32_t mixrounds, uint64_t *prebuf_le, uint8_t *input, uint32_t len, uint8_t *output, int64_t s_cost, uint32_t max_nonce, int thr_id, uint32_t *winning_nonce, uint32_t num_threads, uint32_t *device_target, uint32_t *is_winning, uint32_t num_blocks, uint8_t *sbufs, uint32_t *d_KNonce2);
void update_device_data(int thr_id);

//#define DEBUG
//#define CUDA_DEBUG
//#define CUDA_OUTPUT

//#define DEBUG
//#define DEBUG_CUDA
//#define LOWMEM

int cuda_query() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}

	return nDevices;
}

__constant__ const uint32_t __align__(8) __sha256_init[] = {
	0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
	0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};
__constant__ const uint32_t cpu_K[64] = {
	0x428A2F98U, 0x71374491U, 0xB5C0FBCFU, 0xE9B5DBA5U, 0x3956C25BU, 0x59F111F1U, 0x923F82A4U, 0xAB1C5ED5U,
	0xD807AA98U, 0x12835B01U, 0x243185BEU, 0x550C7DC3U, 0x72BE5D74U, 0x80DEB1FEU, 0x9BDC06A7U, 0xC19BF174U,
	0xE49B69C1U, 0xEFBE4786U, 0x0FC19DC6U, 0x240CA1CCU, 0x2DE92C6FU, 0x4A7484AAU, 0x5CB0A9DCU, 0x76F988DAU,
	0x983E5152U, 0xA831C66DU, 0xB00327C8U, 0xBF597FC7U, 0xC6E00BF3U, 0xD5A79147U, 0x06CA6351U, 0x14292967U,
	0x27B70A85U, 0x2E1B2138U, 0x4D2C6DFCU, 0x53380D13U, 0x650A7354U, 0x766A0ABBU, 0x81C2C92EU, 0x92722C85U,
	0xA2BFE8A1U, 0xA81A664BU, 0xC24B8B70U, 0xC76C51A3U, 0xD192E819U, 0xD6990624U, 0xF40E3585U, 0x106AA070U,
	0x19A4C116U, 0x1E376C08U, 0x2748774CU, 0x34B0BCB5U, 0x391C0CB3U, 0x4ED8AA4AU, 0x5B9CCA4FU, 0x682E6FF3U,
	0x748F82EEU, 0x78A5636FU, 0x84C87814U, 0x8CC70208U, 0x90BEFFFAU, 0xA4506CEBU, 0xBEF9A3F7U, 0xC67178F2U
};


#define PREBUF_LEN 409600
uint64_t host_prebuf_le[20][PREBUF_LEN / 8];
uint8_t host_prebuf_filled[20] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
#define BLOCK_SIZE (32)

uint64_t *device_prebuf_le[20];
uint32_t *device_winning_nonce[20];
uint8_t *device_sbuf[20];
struct hash_state *device_s[20];
uint32_t *device_target[20];
uint32_t *device_is_winning[20];
uint8_t *device_out[20];
uint8_t *device_input[20];
uint8_t *device_sbufs[20];

uint8_t balloon_inited[20] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
uint8_t syncmode_set[20] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
void balloon_cuda_init(int thr_id, uint32_t opt_cuda_syncmode, uint32_t num_threads, uint32_t num_blocks) {
	CUDA_SAFE_CALL(hipSetDevice(thr_id));
	
	if (!syncmode_set[thr_id]) {
		//CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		//CUDA_SAFE_CALL(hipDeviceReset());
		//CUDA_SAFE_CALL(hipSetDeviceFlags(hipDeviceScheduleSpin));
		//CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		//CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		//CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_K), K, sizeof(K), 0, hipMemcpyHostToDevice));
		//hipMemcpyToSymbol(HIP_SYMBOL(c_K), cpu_K, sizeof(cpu_K), 0, hipMemcpyHostToDevice);
		syncmode_set[thr_id] = 1;
	}
#ifdef DEBUG
	printf("DEBUG GPU %d: entering balloon_cuda_init\n", thr_id);
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("%d KB free of total %d KB before init\n", free / 1024, total / 1024);
#endif
	if (!balloon_inited[thr_id]) {
		applog(LOG_INFO, "Initiated GPU #%d", thr_id);
		CUDA_SAFE_CALL(hipMalloc((void**)&device_prebuf_le[thr_id], (PREBUF_LEN / 8) * sizeof(uint64_t)));
		CUDA_SAFE_CALL(hipMalloc((void**)&device_sbuf[thr_id], /*s.n_blocks*/4096 * BLOCK_SIZE));
		CUDA_SAFE_CALL(hipMalloc((void**)&device_is_winning[thr_id], sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc((void**)&device_winning_nonce[thr_id], sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc((void**)&device_s[thr_id], sizeof(struct hash_state)));
		CUDA_SAFE_CALL(hipMalloc((void**)&device_target[thr_id], 8 * sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc((void**)&device_out[thr_id], BLOCK_SIZE * sizeof(uint8_t)));
		CUDA_SAFE_CALL(hipMalloc((void**)&device_input[thr_id], /*len*/80));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_KNonce2[thr_id], 2 * sizeof(uint32_t)));

		
#ifdef LOWMEM
		CUDA_SAFE_CALL(hipMalloc((void**)&device_sbufs[thr_id], num_threads*num_blocks * 4096 * BLOCK_SIZE));
		printf("device_sbufs[thr_id] = %x\n", device_sbufs[thr_id]);
#endif
		balloon_inited[thr_id] = 1;
	}
#ifdef DEBUG
	printf("DEBUG GPU %d: leaving balloon_cuda_init\n", thr_id);
	hipMemGetInfo(&free, &total);
	printf("%d KB free of total %d KB after init\n", free / 1024, total / 1024);
#endif

}

void fill_prebuf(struct hash_state *s, int thr_id) {
#ifdef DEBUG
	printf("DEBUG GPU %d: entering fill_prebuf\n", thr_id);
#endif
	uint8_t host_prebuf[PREBUF_LEN];
	if (!host_prebuf_filled[thr_id]) {
		bitstream_fill_buffer(&s->bstream, host_prebuf, PREBUF_LEN);
		host_prebuf_filled[thr_id] = 1;
		uint8_t *buf = host_prebuf;
		uint64_t *lebuf = host_prebuf_le[thr_id];
		for (int i = 0; i < PREBUF_LEN; i += 8) {
			//bytes_to_littleend8_uint64(buf, lebuf);
			*lebuf <<= 8; *lebuf |= *(buf + 7);
			*lebuf <<= 8; *lebuf |= *(buf + 6);
			*lebuf <<= 8; *lebuf |= *(buf + 5);
			*lebuf <<= 8; *lebuf |= *(buf + 4);
			*lebuf <<= 8; *lebuf |= *(buf + 3);
			*lebuf <<= 8; *lebuf |= *(buf + 2);
			*lebuf <<= 8; *lebuf |= *(buf + 1);
			*lebuf <<= 8; *lebuf |= *(buf + 0);
			*lebuf %= 4096;
			*lebuf <<= 5;
			lebuf++;
			buf += 8;
		}
		update_device_data(thr_id);
	}
#ifdef DEBUG
	printf("DEBUG GPU %d: leaving fill_prebuf\n", thr_id);
#endif
}

void reset_host_prebuf(int thr_id) {
	
		host_prebuf_filled[thr_id] = 0;
	
}


void update_device_data(int thr_id) {
#ifdef DEBUG
	printf("DEBUG GPU %d: entering update_device_data\n", thr_id);
#endif
	CUDA_SAFE_CALL(hipMemcpy(device_prebuf_le[thr_id], host_prebuf_le[thr_id], (PREBUF_LEN / 8) * sizeof(uint64_t), hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(device_prebuf_le), host_prebuf_le, 409600/8 * sizeof(uint64_t), 0, hipMemcpyHostToDevice));
#ifdef DEBUG
	printf("DEBUG GPU %d: leaving update_device_data\n", thr_id);
#endif
}

void balloon_cuda_free(int thr_id) {
	//hipFree(device_prebuf_le[thr_id]);
	//hipFree(device_sbuf[thr_id]);
	//hipFree(device_s[thr_id]);
	//hipFree(device_winning_nonce[thr_id]);
	//hipFree(device_is_winning[thr_id]);
	//hipFree(device_out[thr_id]);
	//hipFree(device_input[thr_id]);
#ifdef LOWMEM
	//hipFree(device_sbufs[thr_id]);
#endif
	//balloon_inited = 0;
}

uint32_t balloon_128_cuda(int thr_id, unsigned char *input, unsigned char *output, uint32_t *target, uint32_t max_nonce, uint32_t num_threads, uint32_t *is_winning, uint32_t num_blocks, uint32_t *h_nounce) {
	return cuda_balloon(thr_id, input, output, 80, 128, 4, target, max_nonce, num_threads, is_winning, num_blocks,h_nounce);
}

//#define NUM_THREADS 256
//#define NUM_THREADS 384
//#define NUM_THREADS 384
//#define NUM_BLOCKS 480
//#define NUM_BLOCKS 48

__host__
uint32_t cuda_balloon(int thr_id, unsigned char *input, unsigned char *output, int32_t len, int64_t s_cost, int32_t t_cost, uint32_t *target, uint32_t max_nonce, uint32_t num_threads, uint32_t *ret_is_winning, uint32_t num_blocks, uint32_t *h_nounce) {
#ifdef DEBUG
	printf("DEBUG GPU %d: entering cuda_balloon\n", thr_id);
#endif

	CUDA_SAFE_CALL(hipSetDevice(thr_id));
	struct balloon_options opts;
	struct hash_state s;
	balloon_init(&opts, s_cost, t_cost);
	hash_state_init(&s, &opts, input);
	fill_prebuf(&s, thr_id);
	uint8_t *pc_sbuf = s.buffer;

#ifdef DEBUG
	if (s.n_blocks > 4096) printf("s.n_blocks = %llu\n", s.n_blocks);
#endif

	uint32_t first_nonce = ((input[76] << 24) | (input[77] << 16) | (input[78] << 8) | input[79]);
	CUDA_SAFE_CALL(hipMemcpy((void**)device_sbuf[thr_id], (void**)s.buffer, s.n_blocks * BLOCK_SIZE, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemsetAsync((void**)d_KNonce2[thr_id], 0xff, 2 * sizeof(uint32_t), gpustream[thr_id]));
	s.buffer = device_sbuf[thr_id];
	CUDA_SAFE_CALL(hipMemcpy((void**)device_s[thr_id], (void**)&s, sizeof(struct hash_state), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy((void**)device_input[thr_id], (void**)input, len, hipMemcpyHostToDevice));
	uint32_t host_winning_nonce = 0;
	uint32_t host_is_winning = 0;
	CUDA_SAFE_CALL(hipMemcpy(device_target[thr_id], target, 8 * sizeof(uint32_t), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy((void**)device_winning_nonce[thr_id], (void**)&host_winning_nonce, sizeof(uint32_t), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy((void**)device_is_winning[thr_id], (void**)&host_is_winning, sizeof(uint32_t), hipMemcpyHostToDevice));
	cudaized_multi << <num_blocks, num_threads, 0, gpustream[thr_id] >> > (device_s[thr_id], t_cost, device_prebuf_le[thr_id], device_input[thr_id], len, device_out[thr_id], s_cost, max_nonce, thr_id, device_winning_nonce[thr_id], num_threads, device_target[thr_id], device_is_winning[thr_id], num_blocks, device_sbufs[thr_id], d_KNonce2[thr_id]);
	CUDA_SAFE_CALL(hipPeekAtLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy((void*)&host_winning_nonce, (void*)device_winning_nonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy((void*)&host_is_winning, (void*)device_is_winning[thr_id],sizeof(uint32_t), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy((void**)h_nounce, (void**)d_KNonce2[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));

#ifdef DEBUG
	if (host_is_winning) {
		printf("[Host (GPU %d)] Winning (%d) nonce: %u\n", thr_id, host_is_winning, host_winning_nonce);
	}
#endif

#ifdef CUDA_OUTPUT
	CUDA_SAFE_CALL(hipMemcpy((void**)output, (void**)device_out[thr_id], BLOCK_SIZE * sizeof(uint8_t), hipMemcpyDeviceToHost));
#endif



	s.buffer = pc_sbuf;
	hash_state_free(&s);

	*ret_is_winning = host_is_winning;
	if (host_is_winning == 0) {
		host_winning_nonce = first_nonce + num_threads*num_blocks - 1;

	}

	return host_winning_nonce;
}

__device__ void * block_index(const struct hash_state *s, size_t i) {
	return s->buffer + (BLOCK_SIZE * i);
}
__device__ void * block_last(const struct hash_state *s) {
	return block_index(s, s->n_blocks - 1);
}
__device__ void cuda_hash_state_extract(const struct hash_state *s, uint8_t out[BLOCK_SIZE]) {
	uint8_t *b = (uint8_t*)block_last(s);
	memcpy((char *)out, (const char *)b, BLOCK_SIZE);
}

//#define CUDA_OUTPUT
__global__ void cudaized_multi(struct hash_state *hs, int32_t mixrounds, uint64_t *prebuf_le, uint8_t *input, uint32_t len, uint8_t *output, int64_t s_cost, uint32_t max_nonce, int thr_id, uint32_t *winning_nonce, uint32_t num_threads, uint32_t *device_target, uint32_t *is_winning, uint32_t num_blocks, uint8_t *sbufs, uint32_t *resNounce) {

	//if (threadIdx.x < 64U) s_K[threadIdx.x] = c_K[threadIdx.x];

#ifdef DEBUG_CUDA
	printf("[Device %d] entering cuda\n", thr_id);
#endif
	uint32_t id = blockDim.x*blockIdx.x + threadIdx.x;
	uint32_t nonce = ((input[76] << 24) | (input[77] << 16) | (input[78] << 8) | input[79]) + id;
	//if (nonce > max_nonce || *is_winning) {
	if (nonce > max_nonce || *is_winning) {

		asm("exit;");
	}
	uint8_t local_input[80];
#ifdef CUDA_OUTPUT
	uint8_t local_output[32];
#endif
	struct hash_state local_s;
	memcpy(local_input, input, len);
	memcpy(&local_s, hs, sizeof(struct hash_state));

#ifdef LOWMEM
	uint8_t *local_sbuf = sbufs + id * 4096 * BLOCK_SIZE;
#else
	uint8_t local_sbuf[4096 * BLOCK_SIZE];
#endif

#ifdef LOWMEM
	memcpy(local_sbuf, hs->buffer, 4096 * BLOCK_SIZE);
#else
	memcpy(&local_sbuf, hs->buffer, 4096 * BLOCK_SIZE);
#endif

	local_s.buffer = local_sbuf;
	((uint32_t*)local_input)[19] = ((nonce & 0xff000000) >> 24) | ((nonce & 0xff0000) >> 8) | ((nonce & 0xff00) << 8) | ((nonce & 0xff) << 24);
	local_s.counter = 0;
	cuda_hash_state_fill(&local_s, local_input, len, mixrounds, s_cost);
	cuda_hash_state_mix(&local_s, mixrounds, prebuf_le);

	if (((uint32_t*)(local_sbuf + (4095 << 5)))[7] < device_target[7]) {

		// Assume winning nonce

		*winning_nonce = nonce;
		*is_winning = 1;

		__threadfence_system();
		asm("exit;");
	}

	}


__device__ void cuda_expand(uint64_t *counter, uint8_t *buf, size_t blocks_in_buf) {
	const uint8_t *blocks[1] = { buf };
	uint8_t *cur = buf + BLOCK_SIZE;

	for (size_t i = 1; i < blocks_in_buf; i++) {
		cuda_compress(counter, cur, blocks,1);
		*blocks += BLOCK_SIZE;
		cur += BLOCK_SIZE;
	}
}



__device__ void cuda_compress(uint64_t *counter, uint8_t *out, const uint8_t *blocks[], size_t blocks_to_comp) {
	uint8_t data[168];
	uint8_t *dp = (uint8_t*)data;
	uint8_t len = BLOCK_SIZE * blocks_to_comp + 8;
	memcpy(dp, counter, 8);
	dp += 8;
	for (unsigned int i = 0; i < blocks_to_comp; i++) {
		memcpy(dp, *(blocks + i), BLOCK_SIZE);
		dp += BLOCK_SIZE;
	}
	device_sha256_generic(data, out, len);
	*counter += 1;
}

__device__ void cuda_hash_state_fill(struct hash_state *s, const uint8_t *in, size_t inlen, int32_t t_cost, int64_t s_cost) {

	
	uint8_t data[132];
	//uint32_t shalen = 8+SALT_LEN+inlen+8+4;
	uint8_t *dp = (uint8_t*)data;
	//if (inlen != 80) {
	//	printf("inlen != 128 (inlen = %d)!!\n", inlen);
	//	if (inlen > 80) inlen = 80;
	//}
	memcpy(dp, &s->counter, 8);
	dp += 8;
	memcpy(dp, in, SALT_LEN);
	dp += SALT_LEN;
	memcpy(dp, in, inlen);
	dp += inlen;
	memcpy(dp, &s_cost, 8);
	dp += 8;
	memcpy(dp, &t_cost, 4);

	device_sha256_generic(data, s->buffer, 132);
	s->counter++;
	cuda_expand(&s->counter, s->buffer, s->n_blocks);
}




__device__ void cuda_hash_state_mix(struct hash_state *s, int32_t mixrounds, uint64_t *prebuf_le) {
	

	
	uint64_t *buf = prebuf_le;
	uint8_t *sbuf = s->buffer;
	const int32_t n_blocks = 4096;
	mixrounds = 4;
	uint8_t *last_block = (sbuf + (BLOCK_SIZE*(n_blocks - 1)));
	uint8_t *blocks[5];
	unsigned char data[8 + BLOCK_SIZE * 5];
	unsigned char *db1 = data + 8;
	unsigned char *db2 = data + 40;
	unsigned char *db3 = data + 72;
	unsigned char *db4 = data + 104;
	unsigned char *db5 = data + 136;
#pragma unroll
	for (int32_t rounds = 0; rounds < mixrounds; rounds++) {
		{ // i = 0
			blocks[0] = last_block;
			blocks[1] = sbuf;
			blocks[2] = (sbuf + ((*(buf++))));
			blocks[3] = (sbuf + ((*(buf++))));
			blocks[4] = (sbuf + ((*(buf++))));

			// New sha256
			//block = (uint8_t**)blocks;
			memcpy(data, &s->counter, 8);
			memcpy(db1, blocks[0], BLOCK_SIZE);
			memcpy(db2, blocks[1], BLOCK_SIZE);
			memcpy(db3, blocks[2], BLOCK_SIZE);
			memcpy(db4, blocks[3], BLOCK_SIZE);
			memcpy(db5, blocks[4], BLOCK_SIZE);
			device_sha256_168byte(data, (uint8_t*)blocks[1]);
			s->counter++;
		}
#pragma unroll
		for (size_t i = 1; i < n_blocks; i++) {
			blocks[0] = blocks[1];
			blocks[1] += BLOCK_SIZE;
			/*blocks[2] = (sbuf + (BLOCK_SIZE * (*(buf++))));
			blocks[3] = (sbuf + (BLOCK_SIZE * (*(buf++))));
			blocks[4] = (sbuf + (BLOCK_SIZE * (*(buf++))));*/

			blocks[2] = (sbuf + ((*(buf++))));
			blocks[3] = (sbuf + ((*(buf++))));
			blocks[4] = (sbuf + ((*(buf++))));

			// New sha256
			memcpy(data, &s->counter, 8);
			memcpy(db1, blocks[0], BLOCK_SIZE);
			memcpy(db2, blocks[1], BLOCK_SIZE);
			memcpy(db3, blocks[2], BLOCK_SIZE);
			memcpy(db4, blocks[3], BLOCK_SIZE);
			memcpy(db5, blocks[4], BLOCK_SIZE);
			device_sha256_168byte(data, (uint8_t*)blocks[1]);
			s->counter++;
		}
		//s->has_mixed = true;
	}
#ifdef DEBUG_CUDA
	if (buf - prebuf_le > 49152) printf("prebuf_le max used: %d, mixrounds = %d, n_blocks = %d\n", buf - prebuf_le, mixrounds, n_blocks);
#endif
}

/*
#define SHA256_CONST(x)         (SHA256_CONST_ ## x)

// constants, as provided in FIPS 180-2 

#define SHA256_CONST_0          0x428a2f98U
#define SHA256_CONST_1          0x71374491U
#define SHA256_CONST_2          0xb5c0fbcfU
#define SHA256_CONST_3          0xe9b5dba5U
#define SHA256_CONST_4          0x3956c25bU
#define SHA256_CONST_5          0x59f111f1U
#define SHA256_CONST_6          0x923f82a4U
#define SHA256_CONST_7          0xab1c5ed5U

#define SHA256_CONST_8          0xd807aa98U
#define SHA256_CONST_9          0x12835b01U
#define SHA256_CONST_10         0x243185beU
#define SHA256_CONST_11         0x550c7dc3U
#define SHA256_CONST_12         0x72be5d74U
#define SHA256_CONST_13         0x80deb1feU
#define SHA256_CONST_14         0x9bdc06a7U
#define SHA256_CONST_15         0xc19bf174U

#define SHA256_CONST_16         0xe49b69c1U
#define SHA256_CONST_17         0xefbe4786U
#define SHA256_CONST_18         0x0fc19dc6U
#define SHA256_CONST_19         0x240ca1ccU
#define SHA256_CONST_20         0x2de92c6fU
#define SHA256_CONST_21         0x4a7484aaU
#define SHA256_CONST_22         0x5cb0a9dcU
#define SHA256_CONST_23         0x76f988daU

#define SHA256_CONST_24         0x983e5152U
#define SHA256_CONST_25         0xa831c66dU
#define SHA256_CONST_26         0xb00327c8U
#define SHA256_CONST_27         0xbf597fc7U
#define SHA256_CONST_28         0xc6e00bf3U
#define SHA256_CONST_29         0xd5a79147U
#define SHA256_CONST_30         0x06ca6351U
#define SHA256_CONST_31         0x14292967U

#define SHA256_CONST_32         0x27b70a85U
#define SHA256_CONST_33         0x2e1b2138U
#define SHA256_CONST_34         0x4d2c6dfcU
#define SHA256_CONST_35         0x53380d13U
#define SHA256_CONST_36         0x650a7354U
#define SHA256_CONST_37         0x766a0abbU
#define SHA256_CONST_38         0x81c2c92eU
#define SHA256_CONST_39         0x92722c85U

#define SHA256_CONST_40         0xa2bfe8a1U
#define SHA256_CONST_41         0xa81a664bU
#define SHA256_CONST_42         0xc24b8b70U
#define SHA256_CONST_43         0xc76c51a3U
#define SHA256_CONST_44         0xd192e819U
#define SHA256_CONST_45         0xd6990624U
#define SHA256_CONST_46         0xf40e3585U
#define SHA256_CONST_47         0x106aa070U

#define SHA256_CONST_48         0x19a4c116U
#define SHA256_CONST_49         0x1e376c08U
#define SHA256_CONST_50         0x2748774cU
#define SHA256_CONST_51         0x34b0bcb5U
#define SHA256_CONST_52         0x391c0cb3U
#define SHA256_CONST_53         0x4ed8aa4aU
#define SHA256_CONST_54         0x5b9cca4fU
#define SHA256_CONST_55         0x682e6ff3U

#define SHA256_CONST_56         0x748f82eeU
#define SHA256_CONST_57         0x78a5636fU
#define SHA256_CONST_58         0x84c87814U
#define SHA256_CONST_59         0x8cc70208U
#define SHA256_CONST_60         0x90befffaU
#define SHA256_CONST_61         0xa4506cebU
#define SHA256_CONST_62         0xbef9a3f7U
#define SHA256_CONST_63         0xc67178f2U
*/

/* Ch and Maj are the basic SHA2 functions. */
#define Ch(b, c, d)     (((b) & (c)) ^ ((~b) & (d)))
#define Maj(b, c, d)    (((b) & (c)) ^ ((b) & (d)) ^ ((c) & (d)))

/* Rotates x right n bits. */
#define ROTR(x, n) __funnelshift_r( (x), (x), (n) )

/* Shift x right n bits */
#define SHR(x, n)       ((x) >> (n))

/* SHA256 Functions */
#define BIGSIGMA0_256(x)        (ROTR((x), 2) ^ ROTR((x), 13) ^ ROTR((x), 22))
#define BIGSIGMA1_256(x)        (ROTR((x), 6) ^ ROTR((x), 11) ^ ROTR((x), 25))
#define SIGMA0_256(x)           (ROTR((x), 7) ^ ROTR((x), 18) ^ SHR((x), 3))
#define SIGMA1_256(x)           (ROTR((x), 17) ^ ROTR((x), 19) ^ SHR((x), 10))

#define	SHA256ROUND(a, b, c, d, e, f, g, h, i, w)			\
T1 = h + BIGSIGMA1_256(e) + Ch(e, f, g) + cpu_K[(i)] + w;	\
d += T1;							\
T2 = BIGSIGMA0_256(a) + Maj(a, b, c);				\
h = T1 + T2

#define        LOAD_BIG_32(addr) (((addr)[0] << 24) | ((addr)[1] << 16) | ((addr)[2] << 8) | (addr)[3])


__device__ void device_sha256_168byte(uint8_t *data, uint8_t *outhash) {
	 __sha256_block_t block[3];
	uint8_t *ptr = (uint8_t*)block;
	memcpy(ptr, data, 168);
	ptr += 168;
	*ptr++ = 0x80;
	memset(ptr, 0, 21);
	ptr += 21;
	*ptr++ = 0x5;
	*ptr++ = 0x40;
	 __sha256_hash_t ohash;
	memcpy(ohash, __sha256_init, 32);
	uint32_t a = ohash[0];
	uint32_t b = ohash[1];
	uint32_t c = ohash[2];
	uint32_t d = ohash[3];
	uint32_t e = ohash[4];
	uint32_t f = ohash[5];
	uint32_t g = ohash[6];
	uint32_t h = ohash[7];
	register uint32_t w0, w1, w2, w3, w4, w5, w6, w7;
	register uint32_t w8, w9, w10, w11, w12, w13, w14, w15;
	register uint32_t T1, T2;
	w0 = LOAD_BIG_32(block[0] + 4 * 0);  SHA256ROUND(a, b, c, d, e, f, g, h, 0, w0);
	w1 = LOAD_BIG_32(block[0] + 4 * 1);  SHA256ROUND(h, a, b, c, d, e, f, g, 1, w1);
	w2 = LOAD_BIG_32(block[0] + 4 * 2);  SHA256ROUND(g, h, a, b, c, d, e, f, 2, w2);
	w3 = LOAD_BIG_32(block[0] + 4 * 3);  SHA256ROUND(f, g, h, a, b, c, d, e, 3, w3);
	w4 = LOAD_BIG_32(block[0] + 4 * 4);  SHA256ROUND(e, f, g, h, a, b, c, d, 4, w4);
	w5 = LOAD_BIG_32(block[0] + 4 * 5);  SHA256ROUND(d, e, f, g, h, a, b, c, 5, w5);
	w6 = LOAD_BIG_32(block[0] + 4 * 6);  SHA256ROUND(c, d, e, f, g, h, a, b, 6, w6);
	w7 = LOAD_BIG_32(block[0] + 4 * 7);  SHA256ROUND(b, c, d, e, f, g, h, a, 7, w7);
	w8 = LOAD_BIG_32(block[0] + 4 * 8);  SHA256ROUND(a, b, c, d, e, f, g, h, 8, w8);
	w9 = LOAD_BIG_32(block[0] + 4 * 9);  SHA256ROUND(h, a, b, c, d, e, f, g, 9, w9);
	w10 = LOAD_BIG_32(block[0] + 4 * 10); SHA256ROUND(g, h, a, b, c, d, e, f, 10, w10);
	w11 = LOAD_BIG_32(block[0] + 4 * 11); SHA256ROUND(f, g, h, a, b, c, d, e, 11, w11);
	w12 = LOAD_BIG_32(block[0] + 4 * 12); SHA256ROUND(e, f, g, h, a, b, c, d, 12, w12);
	w13 = LOAD_BIG_32(block[0] + 4 * 13); SHA256ROUND(d, e, f, g, h, a, b, c, 13, w13);
	w14 = LOAD_BIG_32(block[0] + 4 * 14); SHA256ROUND(c, d, e, f, g, h, a, b, 14, w14);
	w15 = LOAD_BIG_32(block[0] + 4 * 15); SHA256ROUND(b, c, d, e, f, g, h, a, 15, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 16, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 17, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 18, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 19, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 20, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 21, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 22, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 23, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 24, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 25, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 26, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 27, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 28, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 29, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 30, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 31, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 32, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 33, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 34, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 35, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 36, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 37, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 38, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 39, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 40, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 41, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 42, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 43, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 44, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 45, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 46, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 47, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 48, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 49, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 50, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 51, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 52, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 53, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 54, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 55, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 56, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 57, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 58, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 59, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 60, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 61, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 62, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 63, w15);
	a = (ohash[0] += a);
	b = (ohash[1] += b);
	c = (ohash[2] += c);
	d = (ohash[3] += d);
	e = (ohash[4] += e);
	f = (ohash[5] += f);
	g = (ohash[6] += g);
	h = (ohash[7] += h);
	w0 = LOAD_BIG_32(block[1] + 4 * 0);  SHA256ROUND(a, b, c, d, e, f, g, h, 0, w0);
	w1 = LOAD_BIG_32(block[1] + 4 * 1);  SHA256ROUND(h, a, b, c, d, e, f, g, 1, w1);
	w2 = LOAD_BIG_32(block[1] + 4 * 2);  SHA256ROUND(g, h, a, b, c, d, e, f, 2, w2);
	w3 = LOAD_BIG_32(block[1] + 4 * 3);  SHA256ROUND(f, g, h, a, b, c, d, e, 3, w3);
	w4 = LOAD_BIG_32(block[1] + 4 * 4);  SHA256ROUND(e, f, g, h, a, b, c, d, 4, w4);
	w5 = LOAD_BIG_32(block[1] + 4 * 5);  SHA256ROUND(d, e, f, g, h, a, b, c, 5, w5);
	w6 = LOAD_BIG_32(block[1] + 4 * 6);  SHA256ROUND(c, d, e, f, g, h, a, b, 6, w6);
	w7 = LOAD_BIG_32(block[1] + 4 * 7);  SHA256ROUND(b, c, d, e, f, g, h, a, 7, w7);
	w8 = LOAD_BIG_32(block[1] + 4 * 8);  SHA256ROUND(a, b, c, d, e, f, g, h, 8, w8);
	w9 = LOAD_BIG_32(block[1] + 4 * 9);  SHA256ROUND(h, a, b, c, d, e, f, g, 9, w9);
	w10 = LOAD_BIG_32(block[1] + 4 * 10); SHA256ROUND(g, h, a, b, c, d, e, f, 10, w10);
	w11 = LOAD_BIG_32(block[1] + 4 * 11); SHA256ROUND(f, g, h, a, b, c, d, e, 11, w11);
	w12 = LOAD_BIG_32(block[1] + 4 * 12); SHA256ROUND(e, f, g, h, a, b, c, d, 12, w12);
	w13 = LOAD_BIG_32(block[1] + 4 * 13); SHA256ROUND(d, e, f, g, h, a, b, c, 13, w13);
	w14 = LOAD_BIG_32(block[1] + 4 * 14); SHA256ROUND(c, d, e, f, g, h, a, b, 14, w14);
	w15 = LOAD_BIG_32(block[1] + 4 * 15); SHA256ROUND(b, c, d, e, f, g, h, a, 15, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 16, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 17, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 18, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 19, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 20, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 21, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 22, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 23, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 24, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 25, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 26, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 27, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 28, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 29, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 30, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 31, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 32, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 33, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 34, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 35, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 36, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 37, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 38, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 39, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 40, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 41, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 42, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 43, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 44, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 45, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 46, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 47, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 48, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 49, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 50, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 51, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 52, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 53, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 54, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 55, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 56, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 57, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 58, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 59, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 60, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 61, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 62, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 63, w15);
	a = (ohash[0] += a);
	b = (ohash[1] += b);
	c = (ohash[2] += c);
	d = (ohash[3] += d);
	e = (ohash[4] += e);
	f = (ohash[5] += f);
	g = (ohash[6] += g);
	h = (ohash[7] += h);
	w0 = LOAD_BIG_32(block[2] + 4 * 0);  SHA256ROUND(a, b, c, d, e, f, g, h, 0, w0);
	w1 = LOAD_BIG_32(block[2] + 4 * 1);  SHA256ROUND(h, a, b, c, d, e, f, g, 1, w1);
	w2 = LOAD_BIG_32(block[2] + 4 * 2);  SHA256ROUND(g, h, a, b, c, d, e, f, 2, w2);
	w3 = LOAD_BIG_32(block[2] + 4 * 3);  SHA256ROUND(f, g, h, a, b, c, d, e, 3, w3);
	w4 = LOAD_BIG_32(block[2] + 4 * 4);  SHA256ROUND(e, f, g, h, a, b, c, d, 4, w4);
	w5 = LOAD_BIG_32(block[2] + 4 * 5);  SHA256ROUND(d, e, f, g, h, a, b, c, 5, w5);
	w6 = LOAD_BIG_32(block[2] + 4 * 6);  SHA256ROUND(c, d, e, f, g, h, a, b, 6, w6);
	w7 = LOAD_BIG_32(block[2] + 4 * 7);  SHA256ROUND(b, c, d, e, f, g, h, a, 7, w7);
	w8 = LOAD_BIG_32(block[2] + 4 * 8);  SHA256ROUND(a, b, c, d, e, f, g, h, 8, w8);
	w9 = LOAD_BIG_32(block[2] + 4 * 9);  SHA256ROUND(h, a, b, c, d, e, f, g, 9, w9);
	w10 = LOAD_BIG_32(block[2] + 4 * 10); SHA256ROUND(g, h, a, b, c, d, e, f, 10, w10);
	w11 = LOAD_BIG_32(block[2] + 4 * 11); SHA256ROUND(f, g, h, a, b, c, d, e, 11, w11);
	w12 = LOAD_BIG_32(block[2] + 4 * 12); SHA256ROUND(e, f, g, h, a, b, c, d, 12, w12);
	w13 = LOAD_BIG_32(block[2] + 4 * 13); SHA256ROUND(d, e, f, g, h, a, b, c, 13, w13);
	w14 = LOAD_BIG_32(block[2] + 4 * 14); SHA256ROUND(c, d, e, f, g, h, a, b, 14, w14);
	w15 = LOAD_BIG_32(block[2] + 4 * 15); SHA256ROUND(b, c, d, e, f, g, h, a, 15, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 16, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 17, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 18, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 19, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 20, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 21, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 22, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 23, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 24, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 25, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 26, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 27, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 28, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 29, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 30, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 31, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 32, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 33, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 34, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 35, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 36, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 37, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 38, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 39, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 40, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 41, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 42, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 43, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 44, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 45, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 46, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 47, w15);
	w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 48, w0);
	w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 49, w1);
	w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 50, w2);
	w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 51, w3);
	w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 52, w4);
	w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 53, w5);
	w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 54, w6);
	w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 55, w7);
	w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 56, w8);
	w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 57, w9);
	w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 58, w10);
	w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 59, w11);
	w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 60, w12);
	w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 61, w13);
	w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 62, w14);
	w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 63, w15);
	ohash[0] += a;
	ohash[1] += b;
	ohash[2] += c;
	ohash[3] += d;
	ohash[4] += e;
	ohash[5] += f;
	ohash[6] += g;
	ohash[7] += h;
	// finmessy
	uint8_t *h2 = (uint8_t*)ohash;
	uint8_t *outp2 = outhash;
#pragma unroll
	for (int i = 0; i < 32 / 4; i++) {
		// Fix endianness at the same time
		*outp2++ = h2[3];
		*outp2++ = h2[2];
		*outp2++ = h2[1];
		*outp2++ = h2[0];
		h2 += 4;
	}
}

__device__ void device_sha256_generic(uint8_t *data, uint8_t *outhash, uint32_t len) {
	if (len > 184) {
		printf("Longer than 3 blocks (184bytes), sha256_generic not made for this..\n");
		len = 184;
	}
	uint8_t num_blocks = len / 64 + 1;
	uint32_t tot_len = num_blocks * 512 - 65;
	uint32_t num_padding = (tot_len - len * 8) / 8;
	__sha256_block_t block[3];
	uint8_t *ptr = (uint8_t*)block;
	memcpy(ptr, data, len);
	ptr += len;
	*ptr++ = 0x80;
	memset(ptr, 0, num_padding);
	ptr += num_padding;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = 0;
	*ptr++ = ((len * 8) & 0xff00) >> 8;
	*ptr++ = (len * 8) & 0xff;
	__sha256_hash_t ohash;
	memcpy(ohash, __sha256_init, 32);
#pragma unroll
	for (int i = 0; i < num_blocks; i++) {
		uint32_t a = ohash[0];
		uint32_t b = ohash[1];
		uint32_t c = ohash[2];
		uint32_t d = ohash[3];
		uint32_t e = ohash[4];
		uint32_t f = ohash[5];
		uint32_t g = ohash[6];
		uint32_t h = ohash[7];
		uint32_t w0, w1, w2, w3, w4, w5, w6, w7;
		uint32_t w8, w9, w10, w11, w12, w13, w14, w15;
		uint32_t T1, T2;
		w0 = LOAD_BIG_32(block[i] + 4 * 0);  SHA256ROUND(a, b, c, d, e, f, g, h, 0, w0);
		w1 = LOAD_BIG_32(block[i] + 4 * 1);  SHA256ROUND(h, a, b, c, d, e, f, g, 1, w1);
		w2 = LOAD_BIG_32(block[i] + 4 * 2);  SHA256ROUND(g, h, a, b, c, d, e, f, 2, w2);
		w3 = LOAD_BIG_32(block[i] + 4 * 3);  SHA256ROUND(f, g, h, a, b, c, d, e, 3, w3);
		w4 = LOAD_BIG_32(block[i] + 4 * 4);  SHA256ROUND(e, f, g, h, a, b, c, d, 4, w4);
		w5 = LOAD_BIG_32(block[i] + 4 * 5);  SHA256ROUND(d, e, f, g, h, a, b, c, 5, w5);
		w6 = LOAD_BIG_32(block[i] + 4 * 6);  SHA256ROUND(c, d, e, f, g, h, a, b, 6, w6);
		w7 = LOAD_BIG_32(block[i] + 4 * 7);  SHA256ROUND(b, c, d, e, f, g, h, a, 7, w7);
		w8 = LOAD_BIG_32(block[i] + 4 * 8);  SHA256ROUND(a, b, c, d, e, f, g, h, 8, w8);
		w9 = LOAD_BIG_32(block[i] + 4 * 9);  SHA256ROUND(h, a, b, c, d, e, f, g, 9, w9);
		w10 = LOAD_BIG_32(block[i] + 4 * 10); SHA256ROUND(g, h, a, b, c, d, e, f, 10, w10);
		w11 = LOAD_BIG_32(block[i] + 4 * 11); SHA256ROUND(f, g, h, a, b, c, d, e, 11, w11);
		w12 = LOAD_BIG_32(block[i] + 4 * 12); SHA256ROUND(e, f, g, h, a, b, c, d, 12, w12);
		w13 = LOAD_BIG_32(block[i] + 4 * 13); SHA256ROUND(d, e, f, g, h, a, b, c, 13, w13);
		w14 = LOAD_BIG_32(block[i] + 4 * 14); SHA256ROUND(c, d, e, f, g, h, a, b, 14, w14);
		w15 = LOAD_BIG_32(block[i] + 4 * 15); SHA256ROUND(b, c, d, e, f, g, h, a, 15, w15);
		w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 16, w0);
		w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 17, w1);
		w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 18, w2);
		w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 19, w3);
		w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 20, w4);
		w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 21, w5);
		w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 22, w6);
		w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 23, w7);
		w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 24, w8);
		w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 25, w9);
		w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 26, w10);
		w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 27, w11);
		w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 28, w12);
		w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 29, w13);
		w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 30, w14);
		w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 31, w15);
		w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 32, w0);
		w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 33, w1);
		w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 34, w2);
		w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 35, w3);
		w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 36, w4);
		w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 37, w5);
		w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 38, w6);
		w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 39, w7);
		w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 40, w8);
		w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 41, w9);
		w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 42, w10);
		w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 43, w11);
		w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 44, w12);
		w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 45, w13);
		w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 46, w14);
		w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 47, w15);
		w0 = SIGMA1_256(w14) + w9 + SIGMA0_256(w1) + w0; SHA256ROUND(a, b, c, d, e, f, g, h, 48, w0);
		w1 = SIGMA1_256(w15) + w10 + SIGMA0_256(w2) + w1; SHA256ROUND(h, a, b, c, d, e, f, g, 49, w1);
		w2 = SIGMA1_256(w0) + w11 + SIGMA0_256(w3) + w2; SHA256ROUND(g, h, a, b, c, d, e, f, 50, w2);
		w3 = SIGMA1_256(w1) + w12 + SIGMA0_256(w4) + w3; SHA256ROUND(f, g, h, a, b, c, d, e, 51, w3);
		w4 = SIGMA1_256(w2) + w13 + SIGMA0_256(w5) + w4; SHA256ROUND(e, f, g, h, a, b, c, d, 52, w4);
		w5 = SIGMA1_256(w3) + w14 + SIGMA0_256(w6) + w5; SHA256ROUND(d, e, f, g, h, a, b, c, 53, w5);
		w6 = SIGMA1_256(w4) + w15 + SIGMA0_256(w7) + w6; SHA256ROUND(c, d, e, f, g, h, a, b, 54, w6);
		w7 = SIGMA1_256(w5) + w0 + SIGMA0_256(w8) + w7; SHA256ROUND(b, c, d, e, f, g, h, a, 55, w7);
		w8 = SIGMA1_256(w6) + w1 + SIGMA0_256(w9) + w8; SHA256ROUND(a, b, c, d, e, f, g, h, 56, w8);
		w9 = SIGMA1_256(w7) + w2 + SIGMA0_256(w10) + w9; SHA256ROUND(h, a, b, c, d, e, f, g, 57, w9);
		w10 = SIGMA1_256(w8) + w3 + SIGMA0_256(w11) + w10; SHA256ROUND(g, h, a, b, c, d, e, f, 58, w10);
		w11 = SIGMA1_256(w9) + w4 + SIGMA0_256(w12) + w11; SHA256ROUND(f, g, h, a, b, c, d, e, 59, w11);
		w12 = SIGMA1_256(w10) + w5 + SIGMA0_256(w13) + w12; SHA256ROUND(e, f, g, h, a, b, c, d, 60, w12);
		w13 = SIGMA1_256(w11) + w6 + SIGMA0_256(w14) + w13; SHA256ROUND(d, e, f, g, h, a, b, c, 61, w13);
		w14 = SIGMA1_256(w12) + w7 + SIGMA0_256(w15) + w14; SHA256ROUND(c, d, e, f, g, h, a, b, 62, w14);
		w15 = SIGMA1_256(w13) + w8 + SIGMA0_256(w0) + w15; SHA256ROUND(b, c, d, e, f, g, h, a, 63, w15);
		ohash[0] += a;
		ohash[1] += b;
		ohash[2] += c;
		ohash[3] += d;
		ohash[4] += e;
		ohash[5] += f;
		ohash[6] += g;
		ohash[7] += h;
	}

	uint8_t *h = (uint8_t*)ohash;
	uint8_t *outp = outhash;
#pragma unroll
	for (int i = 0; i < 32 / 4; i++) {
		*outp++ = h[3];
		*outp++ = h[2];
		*outp++ = h[1];
		*outp++ = h[0];
		h += 4;
	}
}

